// Copyright (c) 2023, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_<cutlass::float_e4m3_t, 128>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_fp8_hdim128<cutlass::float_e4m3_t>(params, stream);
}
